#include "hip/hip_runtime.h"
#include "util/cuda.hpp"
#include "scene/shallow_water_scene.hpp"

#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace px;

__constant__
scene::ShallowWaterScene::CudaParam_t shallow_water_param[1];

struct CudaShallowWaterProperties
{
    float *u, *v, *h_x, *u_x, *v_x, *h_y, *u_y, *v_y;

    int n_grids = 0;
} sw_prop;

__global__
void initH(unsigned int n, float *h)
{
    PX_CUDA_LOOP(idx, n)
    {
        h[idx] = 1.f;
    }
}

__global__
void shallowWaterDrop(float *h, int seed)
{
    hiprandState_t sd;
    hiprand_init(clock64(), blockIdx.x, 0, &sd);

    auto boundary = 5;
    auto row = std::ceil(hiprand_uniform(&sd) * (shallow_water_param->grid_y-(boundary+boundary))) + boundary;
    auto col = std::ceil(hiprand_uniform(&sd) * (shallow_water_param->grid_x-(boundary+boundary))) + boundary;
    auto sign = hiprand_uniform(&sd) > .5f;

    auto dim = min(shallow_water_param->grid_x, shallow_water_param->grid_y);
    auto height = hiprand_uniform(&sd) + 0.1f;
    auto r = hiprand_uniform(&sd) * dim * .1f + 0.001f;
    auto gap = 1.f / r;
    for (float i = -1; i < 1; i += gap)
    {
        for (float j = -1; j < 1; j += gap)
        {
            auto tar_y = static_cast<int>(row+i*r);
            auto tar_x = static_cast<int>(col+j*r);
            if (tar_y > boundary && tar_y < shallow_water_param->grid_y - boundary
                && tar_x > boundary && tar_x < shallow_water_param->grid_x - boundary)
            {
                auto tar = tar_y*shallow_water_param->grid_x+tar_x;
                if (sign)
                    h[tar] += hiprand_uniform(&sd) * height * expf(-5.f*(i*i+j*j));
                else
                    h[tar] -= hiprand_uniform(&sd) * height * expf(-5.f*(i*i+j*j));
                if (h[tar] < shallow_water_param->height_eps)
                    h[tar] = shallow_water_param->height_eps;
                else if (isnan(h[tar]))
                    h[tar] = 1.f;
            }
        }
    }
}

__global__
void shallowWaterX(unsigned int n, float dt,
                   float *h, float *u, float *v,
                   float *h_x, float *u_x, float *v_x)
{
    PX_CUDA_LOOP(tar, n)
    {
        auto row = tar / shallow_water_param->grid_x;
        auto col = tar % shallow_water_param->grid_x;

        if (row < shallow_water_param->grid_y - 1 &&
            col < shallow_water_param->grid_x - 2)
        {
            auto tar01 = tar + 1;
            auto tar11 = tar01 + shallow_water_param->grid_x;

            h_x[tar] = .5f *
                       ((h[tar11] + h[tar01]) - (u[tar11] - u[tar01]) * dt *
                                                shallow_water_param->inv_gap_x);
            if (h_x[tar] < shallow_water_param->height_eps)
                h_x[tar] = shallow_water_param->height_eps;
            else if (isnan(h_x[tar]))
                h_x[tar] = 1.f;

            u_x[tar] = .5f *
                       ((u[tar11] + u[tar01]) -
                        dt * shallow_water_param->inv_gap_x *
                        (u[tar11] * u[tar11] / h[tar11] -
                         u[tar01] * u[tar01] / h[tar01]
                         + shallow_water_param->half_g *
                           (h[tar11] * h[tar11] - h[tar01] * h[tar01])));
            v_x[tar] = .5f *
                       ((v[tar11] + v[tar01]) -
                        dt * shallow_water_param->inv_gap_x *
                        (u[tar11] * v[tar11] / h[tar11] -
                         u[tar01] * v[tar01] / h[tar01]));
        }

    }
}

__global__
void shallowWaterY(unsigned int n, float dt,
                   float *h, float *u, float *v,
                   float *h_y, float *u_y, float *v_y)
{
    PX_CUDA_LOOP(tar, n)
    {
        auto row = tar / shallow_water_param->grid_x;
        auto col = tar % shallow_water_param->grid_x;

        if (row < shallow_water_param->grid_y - 2 &&
            col < shallow_water_param->grid_x - 1)
        {
            auto tar10 = tar + shallow_water_param->grid_x;
            auto tar11 = tar10 + 1;

            h_y[tar] = .5f *
                       ((h[tar11]+h[tar10]) - (v[tar11]-v[tar10])*dt*shallow_water_param->inv_gap_y);
            if (h_y[tar] < shallow_water_param->height_eps)
                h_y[tar] = shallow_water_param->height_eps;
            else if (isnan(h_y[tar]))
                h_y[tar] = 1.f;

            u_y[tar] = .5f *
                       ((u[tar11]+u[tar10]) - dt*shallow_water_param->inv_gap_y *
                                              (v[tar11]*u[tar11]/h[tar11] - v[tar10]*u[tar10]/h[tar10]));
            v_y[tar] = .5f *
                       ((v[tar11]+v[tar10]) - dt*shallow_water_param->inv_gap_y *
                                              (v[tar11]*v[tar11]/h[tar11] - v[tar10]*v[tar10]/h[tar10]
                                               + shallow_water_param->half_g * (h[tar11]*h[tar11] - h[tar10]*h[tar10])));
        }
    }
}

__global__
void shallowWaterCompose(unsigned int n, float dt,
                         float *h, float *u, float *v,
                         float *h_x, float *u_x, float *v_x,
                         float *h_y, float *u_y, float *v_y)
{
    PX_CUDA_LOOP(tar, n)
    {
        auto row = tar / shallow_water_param->grid_x;
        auto col = tar % shallow_water_param->grid_x;

        if (row > 0 && row < shallow_water_param->grid_y - 2 &&
            col > 0 && col < shallow_water_param->grid_x - 2)
        {
            auto tar0_1 = tar - 1;
            auto tar_1_1 = tar0_1 - shallow_water_param->grid_x;
            auto tar_10 = tar_1_1 + 1;

            h[tar] =  h[tar] -  (dt*shallow_water_param->inv_gap_x) * (u_x[tar0_1] - u_x[tar_1_1])
                             -  (dt*shallow_water_param->inv_gap_y) * (v_y[tar_10] - v_y[tar_1_1]);

//            if (threadIdx.x == 15)
//            {
//                printf("%d")
//            }

            if (h[tar] < shallow_water_param->height_eps)
                h[tar] = shallow_water_param->height_eps;
            else if (isnan(h[tar]))
                h[tar] = 1.f;

            u[tar] -=   (dt*shallow_water_param->inv_gap_x) * (u_x[tar0_1]*u_x[tar0_1]/h_x[tar0_1] - u_x[tar_1_1]*u_x[tar_1_1]/h_x[tar_1_1]
                                          + shallow_water_param->half_g * (h_x[tar0_1]*h_x[tar0_1] - h_x[tar_1_1]*h_x[tar_1_1]))
                        + (dt*shallow_water_param->inv_gap_y) * (v_y[tar_10]*u_y[tar_10]/h_y[tar_10] - v_y[tar_1_1]*u_y[tar_1_1]/h_y[tar_1_1]);
            v[tar] -=   (dt*shallow_water_param->inv_gap_x) * (u_x[tar0_1]*v_x[tar0_1]/h_x[tar0_1] - u_x[tar_1_1]*v_x[tar_1_1]/h_x[tar_1_1])
                        + (dt*shallow_water_param->inv_gap_y) * (v_y[tar_10]*v_y[tar_10]/h_y[tar_10] - v_y[tar_1_1]*v_y[tar_1_1]/h_y[tar_1_1]
                                            + shallow_water_param->half_g * (h_y[tar_10]*h_y[tar_10] - h_y[tar_1_1]*h_y[tar_1_1]));
        }
    }
}

void scene::ShallowWaterScene::cudaInit(void *buffer)
{
    auto n_grids = cuda_param.grid_x * cuda_param.grid_y;
    auto h = reinterpret_cast<float*>(buffer);

    PX_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(shallow_water_param), &cuda_param, sizeof(CudaParam_t), 0,
                                     hipMemcpyHostToDevice));
    if (sw_prop.n_grids != n_grids)
    {
        cudaBufferFree();
        PX_CUDA_CHECK(hipMalloc(&sw_prop.u,   sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.v,   sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.h_x, sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.u_x, sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.v_x, sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.h_y, sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.u_y, sizeof(float)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&sw_prop.v_y, sizeof(float)*n_grids));
        sw_prop.n_grids = n_grids;
    }

    initH<<<cuda::blocks(n_grids), PX_CUDA_THREADS_PER_BLOCK>>>(n_grids, h);

    PX_CUDA_CHECK(hipMemset(sw_prop.u,   0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.v,   0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.h_x, 0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.u_x, 0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.v_x, 0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.h_y, 0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.u_y, 0, sizeof(float)*n_grids));
    PX_CUDA_CHECK(hipMemset(sw_prop.v_y, 0, sizeof(float)*n_grids));

    shallowWaterDrop<<<1, 1>>>(h, n_grids);
}

void scene::ShallowWaterScene::cudaUpdate(void *buffer, float dt,
                                          unsigned int n_iter, float seed)
{
    auto h = reinterpret_cast<float*>(buffer);

    if (seed != 0)
        shallowWaterDrop<<<1, 1>>>(h, seed);

    for (decltype(n_iter) i = 0; i < n_iter; ++i)
    {
        shallowWaterX<<<cuda::blocks(sw_prop.n_grids), PX_CUDA_THREADS_PER_BLOCK>>>
              (sw_prop.n_grids, dt,
                      h, sw_prop.u, sw_prop.v,
                      sw_prop.h_x,  sw_prop.u_x, sw_prop.v_x);
        shallowWaterY<<<cuda::blocks(sw_prop.n_grids), PX_CUDA_THREADS_PER_BLOCK>>>
              (sw_prop.n_grids, dt,
                      h, sw_prop.u, sw_prop.v,
                      sw_prop.h_y,  sw_prop.u_y, sw_prop.v_y);
        shallowWaterCompose<<<cuda::blocks(sw_prop.n_grids), PX_CUDA_THREADS_PER_BLOCK>>>
              (sw_prop.n_grids, dt,
                      h, sw_prop.u, sw_prop.v,
                      sw_prop.h_x,  sw_prop.u_x, sw_prop.v_x,
                      sw_prop.h_y,  sw_prop.u_y, sw_prop.v_y);

        hipblasScopy(cuda_param.grid_y, h+1, cuda_param.grid_x, h, cuda_param.grid_x);
        hipblasScopy(cuda_param.grid_y, sw_prop.u+1, cuda_param.grid_x, sw_prop.u, cuda_param.grid_x);
        hipblasScopy(cuda_param.grid_y, sw_prop.v+1, cuda_param.grid_x, sw_prop.v, cuda_param.grid_x);
        hipblasSscal(cuda_param.grid_y, -1.f, sw_prop.v, cuda_param.grid_x);

        hipblasScopy(cuda_param.grid_y, h+cuda_param.grid_x-2, cuda_param.grid_x, h+cuda_param.grid_x-1, cuda_param.grid_x);
        hipblasScopy(cuda_param.grid_y, sw_prop.u+cuda_param.grid_x-2, cuda_param.grid_x, sw_prop.u+cuda_param.grid_x-1, cuda_param.grid_x);
        hipblasScopy(cuda_param.grid_y, sw_prop.v+cuda_param.grid_x-2, cuda_param.grid_x, sw_prop.v+cuda_param.grid_x-1, cuda_param.grid_x);
        hipblasSscal(cuda_param.grid_y, -1.f, sw_prop.v+cuda_param.grid_x-1, cuda_param.grid_x);

        hipblasScopy(cuda_param.grid_x, h+cuda_param.grid_x, 1, h, 1);
        hipblasScopy(cuda_param.grid_x, sw_prop.u+cuda_param.grid_x, 1, sw_prop.u, 1);
        hipblasScopy(cuda_param.grid_x, sw_prop.v+cuda_param.grid_x, 1, sw_prop.v, 1);
        hipblasSscal(cuda_param.grid_x, -1.f, sw_prop.u, 1);

        hipblasScopy(cuda_param.grid_x, h+cuda_param.grid_x*(cuda_param.grid_y-2), 1, h+cuda_param.grid_x*(cuda_param.grid_y-1), 1);
        hipblasScopy(cuda_param.grid_x, sw_prop.u+cuda_param.grid_x*(cuda_param.grid_y-2), 1, sw_prop.u+cuda_param.grid_x*(cuda_param.grid_y-1), 1);
        hipblasScopy(cuda_param.grid_x, sw_prop.v+cuda_param.grid_x*(cuda_param.grid_y-2), 1, sw_prop.v+cuda_param.grid_x*(cuda_param.grid_y-1), 1);
        hipblasSscal(cuda_param.grid_x, -1.f, sw_prop.u+cuda_param.grid_x*(cuda_param.grid_y-1), 1);
    }
}

void scene::ShallowWaterScene::cudaBufferFree()
{
    if (sw_prop.n_grids == 0)
        return;

    PX_CUDA_CHECK(hipFree(sw_prop.u));
    PX_CUDA_CHECK(hipFree(sw_prop.v));
    PX_CUDA_CHECK(hipFree(sw_prop.h_x));
    PX_CUDA_CHECK(hipFree(sw_prop.u_x));
    PX_CUDA_CHECK(hipFree(sw_prop.v_x));
    PX_CUDA_CHECK(hipFree(sw_prop.h_y));
    PX_CUDA_CHECK(hipFree(sw_prop.u_y));
    PX_CUDA_CHECK(hipFree(sw_prop.v_y));
    sw_prop.n_grids = 0;
}